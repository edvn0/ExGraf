
#include <hip/hip_runtime.h>
__global__ void add(float *A, float *B, float *C, int N) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	int idx = i * N + j; // Compute flattened index
	C[idx] = A[idx] + B[idx];
}
